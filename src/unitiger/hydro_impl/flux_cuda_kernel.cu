
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_HAVE_CUDA

#include <buffer_manager.hpp>
#include <cuda_buffer_util.hpp>
#include "octotiger/options.hpp"
#include "octotiger/cuda_util/cuda_helper.hpp"
#include <cuda_runtime.h>
#include <stream_manager.hpp>

#include "octotiger/unitiger/hydro_impl/flux_kernel_interface.hpp"

#include <hpx/synchronization/once.hpp>

// TODO Duplicated from cell_geometry class - come up with a better way to get this to the device
// Includes flip_dim, faces, xloc and quad_weights
// Move cell geometry class to device?

__device__ inline int flip_dim(const int d, const int flip_dim) {
		int dims[3];
		int k = d;
		for (int dim = 0; dim < 3; dim++) {
			dims[dim] = k % 3;
			k /= 3;
		}
		k = 0;
		dims[flip_dim] = 2 - dims[flip_dim];
		for (int dim = 0; dim < 3; dim++) {
			k *= 3;
			k += dims[2 - dim];
		}
		return k;
}

__device__ const int faces[3][9] = { { 12, 0, 3, 6, 9, 15, 18, 21, 24 }, { 10, 0, 1, 2, 9, 11,
			18, 19, 20 }, { 4, 0, 1, 2, 3, 5, 6, 7, 8 } };

__device__ const int xloc[27][3] = {
	/**/{ -1, -1, -1 }, { +0, -1, -1 }, { +1, -1, -1 },
	/**/{ -1, +0, -1 }, { +0, +0, -1 }, { 1, +0, -1 },
	/**/{ -1, +1, -1 }, { +0, +1, -1 }, { +1, +1, -1 },
	/**/{ -1, -1, +0 }, { +0, -1, +0 }, { +1, -1, +0 },
	/**/{ -1, +0, +0 }, { +0, +0, +0 }, { +1, +0, +0 },
	/**/{ -1, +1, +0 }, { +0, +1, +0 }, { +1, +1, +0 },
	/**/{ -1, -1, +1 }, { +0, -1, +1 }, { +1, -1, +1 },
	/**/{ -1, +0, +1 }, { +0, +0, +1 }, { +1, +0, +1 },
	/**/{ -1, +1, +1 }, { +0, +1, +1 }, { +1, +1, +1 } };

__device__ const double quad_weights[9] = { 16. / 36., 1. / 36., 4. / 36., 1. / 36., 4. / 36., 4.
			/ 36., 1. / 36., 4. / 36., 1. / 36. };

hpx::lcos::local::once_flag flag1;

__host__ void init_gpu_masks(bool *masks) {
  auto masks_boost = create_masks();
  cudaMemcpy(masks, masks_boost.data(), NDIM * 1000 * sizeof(bool), cudaMemcpyHostToDevice);
}

__host__ const bool* get_gpu_masks(void) {
    // TODO Create class to handle these read-only, created-once GPU buffers for masks. This is a reoccuring problem
    static bool *masks = recycler::recycle_allocator_cuda_device<bool>{}.allocate(NDIM * 1000);
    hpx::lcos::local::call_once(flag1, init_gpu_masks, masks);
    return masks;
}

__device__ const int offset = 0;
__device__ const int compressedH_DN[3] = {100, 10, 1};
__device__ const int face_offset = 27 * 1000;
__device__ const int dim_offset = 1000;

__global__ void
__launch_bounds__(128, 2)
 flux_cuda_kernel(const double * __restrict__ q_combined, const double * __restrict__ x_combined, double * __restrict__ f_combined,
    double * amax, int * amax_indices, int * amax_d, const bool * __restrict__ masks, const double omega, const double dx, const double A_, const double B_, const double fgamma, const double de_switch_1) {
  __shared__ double sm_amax[128];
  __shared__ int sm_d[128];
  __shared__ int sm_i[128];

  const int nf = 15;

  double local_f[15] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  double local_x[3] = {0.0, 0.0, 0.0};
  double local_vg[3] = {0.0, 0.0, 0.0};

  double current_amax = 0.0;
  int current_d = 0;

  // 3 dim 1000 i workitems
  const int dim = blockIdx.z;
  const int tid = threadIdx.x * 64 + threadIdx.y * 8 + threadIdx.z;
  const int index = blockIdx.y * 128 + tid + 104;
  for (int f = 0; f < nf; f++) {
      f_combined[dim * 15 * 1000 + f * 1000 + index] = 0.0;
  }
  if (index < 1000) {
    double mask = masks[index + dim * dim_offset];
    if(mask != 0.0) {
      for (int fi = 0; fi < 9; fi++) {    // 9
        double this_ap = 0.0, this_am = 0.0;    // tmps
        const int d = faces[dim][fi];
        const int flipped_dim = flip_dim(d, dim);
        for (int dim = 0; dim < 3; dim++) {
            local_x[dim] = x_combined[dim * 1000 + index] + (0.5 * xloc[d][dim] * dx);
        }
        local_vg[0] = -omega * (x_combined[1000 + index] + 0.5 * xloc[d][1] * dx);
        local_vg[1] = +omega * (x_combined[index] + 0.5 * xloc[d][0] * dx);
        local_vg[2] = 0.0;
        inner_flux_loop2<double>(omega, nf, A_, B_, q_combined, local_f, local_x, local_vg,
          this_ap, this_am, dim, d, dx, fgamma, de_switch_1,
          dim_offset * d + index, dim_offset * flipped_dim - compressedH_DN[dim] + index, face_offset);
        this_ap *= mask;
        this_am *= mask;
        const double amax_tmp = max_wrapper(this_ap, (-this_am));
        if (amax_tmp > current_amax) {
          current_amax = amax_tmp;
          current_d = d;
        }
        for (int f = 1; f < nf; f++) {
          f_combined[dim * 15 * 1000 + f * 1000 + index] += quad_weights[fi] * local_f[f];
        }
      }
    }
    for (int f = 10; f < nf; f++) {
      f_combined[dim * 15 * 1000 + index] += f_combined[dim * 15 * 1000 + f * 1000 + index];
    }
  }
  // Find maximum:
  sm_amax[tid] = current_amax;
  sm_d[tid] = current_d;
  sm_i[tid] = index;
  __syncthreads();
  // Max reduction with multiple warps
  for (int tid_border = 64; tid_border >= 32; tid_border /= 2) {
    if(tid < tid_border) {
      if (sm_amax[tid + tid_border] > sm_amax[tid]) {
        sm_amax[tid] = sm_amax[tid + tid_border];
        sm_d[tid] = sm_d[tid + tid_border];
        sm_i[tid] = sm_i[tid + tid_border];
      }
    }
    __syncthreads();
  }
  // Max reduction within one warps
  for (int tid_border = 16; tid_border >= 1; tid_border /= 2) {
  if(tid < tid_border) {
      if (sm_amax[tid + tid_border] > sm_amax[tid]) {
        sm_amax[tid] = sm_amax[tid + tid_border];
        sm_d[tid] = sm_d[tid + tid_border];
        sm_i[tid] = sm_i[tid + tid_border];
      }
    }
  }

  if (tid == 0) {
    //printf("Block %i %i TID %i %i \n", blockIdx.y, blockIdx.z, tid, index);
    const int block_id = blockIdx.y + dim * 7;
    amax[block_id] = sm_amax[0];
    amax_indices[block_id] = sm_i[0];
    amax_d[block_id] = sm_d[0];

    // Save face to the end of the amax buffer
    const int flipped_dim = flip_dim(sm_d[0], dim);
    for (int f = 0; f < nf; f++) {
      amax[21 + block_id * 30 + f] = q_combined[sm_i[0] + f * face_offset + dim_offset * sm_d[0]];
      amax[21 + block_id * 30 + 15 + f] = q_combined[sm_i[0] - compressedH_DN[dim] + f * face_offset +
          dim_offset * flipped_dim];
    }
  }
  return;
}

timestep_t launch_flux_cuda(stream_interface<hpx::cuda::experimental::cuda_executor, pool_strategy>& executor,
    double* device_q,
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_f,
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_x, double* device_x,
    safe_real omega, const size_t nf_, double dx, size_t device_id) {
    timestep_t ts;
    const cell_geometry<3, 8> geo;

    recycler::cuda_device_buffer<double> device_f(NDIM * 15 * 1000 + 32, device_id);
    const bool *masks = get_gpu_masks();

    recycler::cuda_device_buffer<double> device_amax(7 * NDIM * (1 + 2 * 15));
    recycler::cuda_device_buffer<int> device_amax_indices(7 * NDIM);
    recycler::cuda_device_buffer<int> device_amax_d(7 * NDIM);
    double A_ = physics<NDIM>::A_;
    double B_ = physics<NDIM>::B_;
    double fgamma = physics<NDIM>::fgamma_;
    double de_switch_1 = physics<NDIM>::de_switch_1;

    dim3 const grid_spec(1, 7, 3);
    dim3 const threads_per_block(2, 8, 8);
    void* args[] = {&(device_q),
      &(device_x), &(device_f.device_side_buffer), &(device_amax.device_side_buffer),
      &(device_amax_indices.device_side_buffer), &(device_amax_d.device_side_buffer),
      &masks, &omega, &dx, &A_, &B_, &fgamma, &de_switch_1};
    executor.post(
    cudaLaunchKernel<decltype(flux_cuda_kernel)>,
    flux_cuda_kernel, grid_spec, threads_per_block, args, 0);

    // Move data to host
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> amax(7 * NDIM * (1 + 2 * 15));
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_indices(7 * NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_d(7 * NDIM);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax.data(),
               device_amax.device_side_buffer, (7 * NDIM * (1 + 2 * 15)) * sizeof(double),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_indices.data(),
               device_amax_indices.device_side_buffer, 7 * NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_d.data(),
               device_amax_d.device_side_buffer, 7 * NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    auto fut = hpx::async(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, combined_f.data(), device_f.device_side_buffer,
               (NDIM * 15 * 1000 + 32) * sizeof(double), cudaMemcpyDeviceToHost);
    fut.get();

    // Find Maximum
    size_t current_dim = 0;
    for (size_t dim_i = 1; dim_i < 7 * NDIM; dim_i++) {
      if (amax[dim_i] > amax[current_dim]) { 
        current_dim = dim_i;
      }
    }
    std::vector<double> URs(nf_), ULs(nf_);
    const size_t current_max_index = amax_indices[current_dim];
    const size_t current_d = amax_d[current_dim];
    ts.a = amax[current_dim];
    ts.x = combined_x[current_max_index];
    ts.y = combined_x[current_max_index + 1000];
    ts.z = combined_x[current_max_index + 2000];
    const size_t current_i = current_dim;
    current_dim = current_dim / 7;
    const auto flipped_dim = geo.flip_dim(current_d, current_dim);
    constexpr int compressedH_DN[3] = {100, 10, 1};
    for (int f = 0; f < nf_; f++) {
        URs[f] = amax[21 + current_i * 30 + f];
        ULs[f] = amax[21 + current_i * 30 + 15 + f];
    }
    ts.ul = std::move(ULs);
    ts.ur = std::move(URs);
    ts.dim = current_dim;
    return ts;
}


#endif
